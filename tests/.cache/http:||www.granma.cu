#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es">

<head prefix="og: http://ogp.me/ns#">
    <meta charset="utf-8">
	<meta name="google-site-verification" content="y_GTvuIlKGzzKjky35SOfpzPwzpBpGsEpE7bwkz0XP8" /> 

    <meta name="viewport" content="width=device-width, initial-scale=1.0">
    <meta name="description" content="Órgano Oficial del Comité Central del Partido Comunista de Cuba. Fundado el 3 de octubre de 1965. Actualización de noticias 24 horas.">
	<meta name="keywords" content="Cuba, Noticias, Diario Granma, Órgano oficial, Revolución Cubana, Fidel Castro, Raúl Castro, Izquierda">
	<link rel="canonical" href="http://www.granma.cu/">
    <base href="http://www.granma.cu/" />
	<meta >

    <link rel="shortcut icon" href="/static/ico/favicon.png">
  

  <!-- Etiqueta global de sitio (gtag.js) de Google Analytics -->

	
	
	    <title>Granma - Órgano oficial del PCC</title>
        <style>
        @font-face {
            font-family: 'Amplitude Medium';
            src: url('/static/fonts/amplitudemedium-webfont.eot');
            src: url('/static/fonts/amplitudemedium-webfont.eot?#iefix') format('embedded-opentype'),
            url('/static/fonts/amplitudemedium-webfont.woff') format('woff'),
            url('/static/fonts/amplitudemedium-webfont.ttf') format('truetype'),
            url('/static/fonts/amplitudemedium-webfont.svg#Amplitude Medium') format('svg');
            font-weight: normal;
            font-style: normal;

        }

        @font-face {
            font-family: 'amplitude-regularregular';
            src: url('/static/fonts/amplituderegular-webfont.eot');
            src: url('/static/fonts/amplituderegular-webfont.eot?#iefix') format('embedded-opentype'),
            url('/static/fonts/amplituderegular-webfont.woff') format('woff'),
            url('/static/fonts/amplituderegular-webfont.ttf') format('truetype'),
            url('/static/fonts/amplituderegular-webfont.svg#amplitude-regularregular') format('svg');
            font-weight: normal;
            font-style: normal;

        }

        @font-face {
            font-family: 'Open Sans';
            src: url('/static/fonts/opensans-webfont.eot');
            src: url('/static/fonts/opensans-webfont.eot?#iefix') format('embedded-opentype'),
            url('/static/fonts/opensans-webfont.woff') format('woff'),
            url('/static/fonts/opensans-webfont.ttf') format('truetype'),
            url('/static/fonts/opensans-webfont.svg#Open Sans') format('svg');
            font-weight: normal;
            font-style: normal;

        }

    </style>
    <!-- Bootstrap core CSS -->
    <link href="/static/css/bootstrap.min.css" rel="stylesheet">
	
    <link href="/static/css/infografia.css" rel="stylesheet">
    <!-- Custom styles for this template -->
    <link href="/static/css/styles.css" rel="stylesheet">
	
	    <link rel="stylesheet" href="/static/css/linea/stylelinea.css">


    <link rel="stylesheet" type="text/css" href="/static/css/bar-porcent.css">

    
    <!-- HTML5 shim and Respond.js IE8 support of HTML5 elements and media queries -->
<!--[if lt IE 9]>
<script src="/static/js/html5shiv.js"></script>
<script src="/static/js/respond.min.js"></script>
<![endif]-->




    <!-- BEGIN Open Graph tags -->
    <meta property='og:type' content='article' />
    <meta property='og:title' content="Granma - Órgano oficial del PCC"/>
    <meta property="og:site_name" content="Granma.cu"/>
    <meta property='og:url' content="http://www.granma.cu" />
    <meta property="og:description" content="Órgano Oficial del Comité Central del Partido Comunista de Cuba. Fundado el 3 de octubre de 1965. Actualización de noticias 24 horas."/>
	<meta property='og:image' content='http://www.granma.cu//static/img/logo-facebook.jpg' />
    <meta property='og:locale' content='es_ES'/>
<!-- END Open Graph tags -->

<!-- BEGIN Twitter Card -->
    <meta name="twitter:card" content="summary_large_image" />
    <meta name="twitter:site" content="@Granma_Digital" />
    <meta name="twitter:title" content="Granma - Órgano oficial del PCC" />
    <meta name="twitter:description" content="Órgano Oficial del Comité Central del Partido Comunista de Cuba. Fundado el 3 de octubre de 1965. Actualización de noticias 24 horas." />
    <meta name="twitter:image" content="http://www.granma.cu//static/img/logo-facebook.jpg" />
    <meta name="twitter:url" content="http://www.granma.cu" />
   
<!-- END Twitter Card -->

</head>

<body>
    <div class="container">
        <!-- Static navbar -->
        <div class="navbar navbar-default navbar-static-top g-top-menu g-menus">
            <div class="navbar-header">
                <button type="button" class="navbar-toggle" data-toggle="collapse" data-target=".navbar-collapse-top">
                    <span class="language-globe"></span>
                </button>
                <a class="navbar-brand" href="/multimedia">MULTIMEDIA</a><span class="navbar-brand">|</span>
                <a class="navbar-brand" href="/blogs">BLOGS</a><span class="navbar-brand">|</span>
                <a class="navbar-brand" href="/impreso">EDICI&Oacute;N IMPRESA</a><span class="navbar-brand">|</span>
                 <a class="navbar-brand" href="/webviejo">SITIO WEB DE AÑOS ANTERIORES</a><span class="navbar-brand"></span>
				
                <!--a class="navbar-brand" href="/impresosesp">IMPRESOS ESPECIALES</a-->
            </div>
            <div class="navbar-collapse collapse navbar-collapse-top">
                <ul class="nav navbar-nav navbar-right">
                    <li class="text-right"><a target="_blank" href="http://en.granma.cu">English</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://fr.granma.cu">Fran&ccedil;ais</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://de.granma.cu">Deutsch</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://pt.granma.cu">Portugu&ecirc;s</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://it.granma.cu">Italiano</a></li>
                </ul>
            </div><!--/.nav-collapse -->
        </div>
        <div class="row g-mid-header">
            <div class="col-md-3 col-sm-3 g-header-logo"><a href="/"><img src="/static/img/granma-logo.png" class="img-responsive" alt="Responsive image"></a></div>
            <div class="col-md-2 col-sm-2 hidden-xs g-header-heroes"><img src="/static/img/heroes-cabezal.png" class="img-responsive" alt="Responsive image"></div>
            <div class="col-md-3 col-sm-3 g-header-titles"><p class="g-mid-header-date hidden-xs">LA HABANA, 23 DE MAYO DE 2020</p><h5 class="g-mid-header-subtitle">&Oacute;RGANO OFICIAL DEL COMIT&Eacute; CENTRAL<br class="hidden-xs"/> DEL PARTIDO COMUNISTA DE CUBA</h5></div>
            <div class="col-md-4 col-sm-4 navbar-right g-header-timesocial hidden-xs">
                <div class=""><p class="g-header-timetitle text-right">ÚLTIMA ACTUALIZACIÓN</p><p class="g-header-time text-right">  15:36:45</p></div>
                <div class="pull-right g-header-social">
                    <a target="_blank" href="https://www.facebook.com/granmadigital" class="g-mid-header-fb"><span class="sr-only">facebook</span></a>
                    <a target="_blank" href="https://twitter.com/Granma_Digital" class="g-mid-header-tw"><span class="sr-only">twitter</span></a>
                    <a target="_blank" href="https://instagram.com/granmadigital" class="g-mid-header-ig"><span class="sr-only">instagram</span></a>
                    <a target="_blank" href="https://youtube.com/channel/UCCXe7yEuB8qy34HiRLp7I4A" class="g-mid-header-yt"><span class="sr-only">youtube</span></a>
                    <a target="_blank" href="http://t.me/periodicogranma" class="g-mid-header-tg"><span class="sr-only">telegram</span></a>
                    <a target="_blank" href="/feed" class="g-mid-header-rss"><span class="sr-only">rss</span></a>
                </div>
            </div>
        </div>

        <nav class="navbar navbar-default g-menus g-main-menu" role="navigation">
  <!-- Brand and toggle get grouped for better mobile display -->
  <div class="navbar-header">
    <button type="button" class="navbar-toggle" data-toggle="collapse" data-target=".navbar-main-collapse">
      <span class="sr-only">Toggle navigation</span>
      <span class="icon-bar"></span>
      <span class="icon-bar"></span>
      <span class="icon-bar"></span>
    </button>
    <a class="navbar-brand visible-xs" href="#">Portada</a>
  </div>

  <!-- Collect the nav links, forms, and other content for toggling -->
  <div class="collapse navbar-collapse navbar-main-collapse">

    <ul class="nav navbar-nav">    
      <li class="active"><a href="/">Portada</a></li>
          <li ><a href="/cuba">Cuba</a></li>
          <li ><a href="/mundo">Mundo</a></li>
          <li ><a href="/deportes">Deportes</a></li>
          <li ><a href="/cultura">Cultura</a></li>
          <li ><a href="/opinion">Opinión</a></li>
          <li ><a href="/ciencia">Ciencia</a></li>
          <li ><a href="/salud">Salud</a></li>
    	
					<li class="dropdown"><a class="dropdown-toggle" data-toggle="dropdown" href="javascript:void(0);" aria-expanded="false">ESPECIALES ▼ </a>
			        <ul class="dropdown-menu">
			          			            <!--li ><a href="/cuba-covid-19">Cuba frente a la Covid-19</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/cuba-covid-19">Cuba frente a la Covid-19</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/conexion-toxio-2020">Conexión Tokio 2020</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/conexion-toxio-2020">Conexión Tokio 2020</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/panamericanos-lima-2019">Panamericanos Lima 2019</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/panamericanos-lima-2019">Panamericanos Lima 2019</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/gracias-por-la-vida">Más que médicos</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/gracias-por-la-vida">Más que médicos</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/reforma-constitucional">Reforma Constitucional en Cuba</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/reforma-constitucional">Reforma Constitucional en Cuba</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/juegos-centroamericanos">Barranquilla 2018</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/juegos-centroamericanos">Barranquilla 2018</a></li>
						<li class="divider" ></li>
			          					<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/portadas-especiales">VER MAS...</a></li>

			        </ul>
			      </li>
    </ul>

    <form action="/archivo" class="navbar-form navbar-right" role="search">
     <div class="">
      <div class="input-group">
        <input name="q" type="search" class="form-control" placeholder="Buscar" value="">
        <span class="input-group-btn">
          <button class="btn btn-default" type="submit"><span class="glyphicon glyphicon-search"></span></button>
        </span>
      </div><!-- /input-group -->
    </div><!-- /.col-lg-6 -->

  </form>

</div><!-- /.navbar-collapse -->
</nav>
        
<div class="row g-middle-container">

    <div>
    <div class="row g-middle-container g-big-story2">
    <div class="col-md-8 container-left g-big-story-main">
        <article>
            <h2><a href="cuba/2020-05-22/arriba-a-cuba-vuelo-charter-procedente-de-estados-unidos-con-ciudadanos-varados">Cuba no deja desamparado a ninguno de sus hijos</a></h2>
            <figure>
                <a href="cuba/2020-05-22/arriba-a-cuba-vuelo-charter-procedente-de-estados-unidos-con-ciudadanos-varados"><img src="/file/img/2020/05/medium/f0169989.jpg" class="img-responsive" alt="Viajeros de cubanos varados en EE.UU. llegan a cuba"></a>
            </figure>
            </br>

            <p>Las autoridades cubanas mantienen un seguimiento permanente a la situación de nuestros connacionales varados en diferentes países, al tiempo que se realizan esfuerzos para, siempre que existan posibilidades, concretar su retorno a nuestro país</p>
            <p class="g-story-comments"><a href="cuba/2020-05-22/arriba-a-cuba-vuelo-charter-procedente-de-estados-unidos-con-ciudadanos-varados#comentarios">5 COMENTARIOS</a></p>
        
		    
		                    </article>
    </div><!--/big-->

    <div class="col-md-4 container-right g-regular-story g-big-story-related">
        <article class="g-regular-story">
            <h3 class="g-story-epi"></h3>
            <h2><a href="mundo/2020-05-23/quien-no-quiere-la-paz-rine-con-la-diplomacia-23-05-2020-00-05-04">Quien no quiere la paz, riñe con la diplomacia</a></h2>
                    <p>La actitud del Gobierno colombiano de desconocer el protocolo de Ruptura del proceso de paz y exigir a Cuba la captura y extradición de los miembros de la delegación del ELN, es una violación del acuerdo suscrito por el Estado colombiano con seis países garantes; un precedente muy grave que pone en riesgo la búsqueda de soluciones negociadas en el mundo</p>

        </article>

    </div>

    <div class="col-md-4 container-right g-regular-story g-big-story-related">
        <article class="g-regular-story">
            <h3 class="g-story-epi"></h3>
            <h2><a href="cuba-covid-19/2020-05-22/ningun-nino-cubano-presenta-el-sindrome-respiratorio-que-advierte-la-oms-21-05-2020-15-05-17">Ningún niño cubano presenta el síndrome respiratorio que advierte la OMS</a></h2>
                            <figure class="col-md-6 g-story-img">
                    <img src="/file/img/2020/05/thumb/f0170007.jpg" class="img-responsive" alt="Viajeros de cubanos varados en EE.UU. llegan a cuba">
                </figure>
                        <p>Ninguno de los niños en Cuba, reportados como positivos a la COVID-19, presenta el síndrome respiratorio, que recientemente ha alertado la Organización Mundial de la Salud (OMS), ni ninguna otra complicación, informó el doctor Francisco Durán, director nacional de Epidemiología del Ministerio de Salud Pública (Minsap)</p>

        </article>

    </div>

    <div class="col-md-4 container-right g-regular-story g-big-story-related">
        <article class="g-regular-story">
            <h3 class="g-story-epi">cuba y áfrica</h3>
            <h2><a href="mundo/2020-05-23/las-huellas-imborrables-de-los-que-caminaron-juntos-23-05-2020-01-05-34">Las huellas imborrables de los que caminaron juntos</a></h2>
                            <figure class="col-md-6 g-story-img">
                    <img src="/file/img/2020/05/thumb/f0170005.jpg" class="img-responsive" alt="Médicos cubanos atienden a niños africanos.">
                </figure>
                        <p>En ocasión del Día de África, la diplomática Gisela García, directora de África Subsahariana del Ministerio de Relaciones Exteriores de la República de Cuba, concedió una entrevista a Granma</p>

        </article>

    </div>
    <div class="col-md-12 big-story2-line"></div>
</div></div>

    <div class="col-md-8">

        <div>
    <div class="col-md-12 g-big-story">
    <figure class="col-md-6 g-story-img">
        <a href="cuba-covid-19/2020-05-23/en-medio-del-enfrentamiento-a-la-covid-19-el-pais-ha-mantenido-su-vitalidad-23-05-2020-00-05-26"><img src="/file/img/2020/05/thumb/f0169991.jpg" class="img-responsive" alt="Las labores del Instituto Nacional de Recursos Hidráulicos se han concentrado fundamentalmente en obras que permiten dar valor de uso a instalaciones 
y disminuir los efectos de la sequía. Foto: José Manuel Correa " ></a>
    </figure>
    <h3 class="g-story-epi"></h3>
    <h2><a href="cuba-covid-19/2020-05-23/en-medio-del-enfrentamiento-a-la-covid-19-el-pais-ha-mantenido-su-vitalidad-23-05-2020-00-05-26">En medio del enfrentamiento a la COVID-19 el país ha mantenido su vitalidad</a></h2>
    <p>Bajo el precepto defendido por el Gobierno cubano de que nada puede entorpecer el empeño de continuar salvando vidas en la batalla contra la COVID-19, el Presidente de la República, Miguel Díaz-Canel Bermúdez, y el primer ministro, Manuel Marrero Cruz, dieron continuidad este viernes a la habitual reunión con el grupo temporal de trabajo para enfrentar el nuevo coronavirus, donde se ratificó que, a pesar del complejo escenario y las medidas adoptadas, el país ha mantenido su vitalidad</p>    <p class="g-story-comments"><a href="cuba-covid-19/2020-05-23/en-medio-del-enfrentamiento-a-la-covid-19-el-pais-ha-mantenido-su-vitalidad-23-05-2020-00-05-26#comentarios">1 COMENTARIO</a></p>
    </div><!--/big--></div>
        <div id="column_0" class="col-md-6 col-sm-6 g-regular-story container-left">

                    <article id="story_4526719" data-id="4526719" data-image="170037">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-23/recibe-brigada-medica-cubana-en-lombardia-un-emocionante-homenaje-fotos-video-23-05-2020-15-05-07">Recibe brigada médica cubana en Lombardía un emocionante homenaje (+ Fotos) (+Video)</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0170037.jpg" class="img-responsive" alt="covid" >
    </figure>
    <div class="sumario">
        <p>Los 36 médicos, 15 enfermeros y un especialista en logística recibieron el reconocimiento en la explanada, puestos de pie, con mascarillas de protección facial y respetando la distancia de seguridad establecida para evitar el contagio</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-23/recibe-brigada-medica-cubana-en-lombardia-un-emocionante-homenaje-fotos-video-23-05-2020-15-05-07#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526711" data-id="4526711" data-image="170013">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-23/realizan-desinfeccion-en-hospital-camagueyano-amalia-simoni-para-normalizar-sus-servicios-23-05-2020-10-05-01">Realizan desinfección en hospital camagüeyano Amalia Simoni para normalizar sus servicios</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0170013.jpg" class="img-responsive" alt="hospital" >
    </figure>
    <div class="sumario">
        <p>Tras haberse habilitado para tratamiento de casos positivos y sospechosos de la COVID-19, el centro reiniciará la atención a todo el pueblo al no contar ya con casos positivos ingresados</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-23/realizan-desinfeccion-en-hospital-camagueyano-amalia-simoni-para-normalizar-sus-servicios-23-05-2020-10-05-01#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526715" data-id="4526715" data-image="166903">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-23/preve-rusia-comenzar-produccion-de-su-vacuna-contra-la-covid-19-en-agosto-23-05-2020-11-05-06">Prevé Rusia comenzar producción de su vacuna contra la COVID-19 en agosto</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/04/thumb/f0166903.jpg" class="img-responsive" alt="Vacunas" >
    </figure>
    <div class="sumario">
        <p>El director del Instituto Nikolái Gamaleya, Alexandr Guintsburg, señaló que los ensayos preliminares de la vacuna en animales mostraron buenos resultados</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-23/preve-rusia-comenzar-produccion-de-su-vacuna-contra-la-covid-19-en-agosto-23-05-2020-11-05-06#comentarios">1 COMENTARIO</a></p>
			</article>                    <article id="story_4526717" data-id="4526717" data-image="170031">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-23/programa-copa-airlines-reiniciar-vuelos-el-26-de-junio-23-05-2020-11-05-36">Programa Copa Airlines reiniciar vuelos el 26 de junio</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0170031.jpg" class="img-responsive" alt="Copa" >
    </figure>
    <div class="sumario">
        <p>El presidente del holding aéreo, Pedro Heilbron, dijo que debido a la baja demanda que se registrará en los próximos meses, la compañía solo usará al principio el 10 por ciento de su flota y en la medida que se amplíen los viajeros prevén que para diciembre estarían usando menos de la mitad de sus aviones</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-23/programa-copa-airlines-reiniciar-vuelos-el-26-de-junio-23-05-2020-11-05-36#comentarios">1 COMENTARIO</a></p>
			</article>                    <article id="story_4526675" data-id="4526675" data-image="169981">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-22/hay-un-coronamundo-anterior-al-coronavirus-22-05-2020-23-05-04">Hay un coronamundo anterior al coronavirus</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169981.jpg" class="img-responsive" alt="Dormitorio que atiende en Turín a mujeres de varias nacionalidades que son «personas en situación de calle». Foto del autor" >
    </figure>
    <div class="sumario">
        <p>Lo cierto es que Italia recibió ayuda de países que tienen mucho menos, como Cuba y Albania, un país vecino pobre. Mientras que países ricos y vecinos como Francia y Alemania cerraron la frontera y se aislaron, los cubanos atravesaron el Océano para venir a ayudarnos</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-22/hay-un-coronamundo-anterior-al-coronavirus-22-05-2020-23-05-04#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526667" data-id="4526667" data-image="169969">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-22/agradezco-haber-nacido-en-cuba-de-lo-contrario-no-hubiera-salvado-mi-vida-22-05-2020-22-05-09">Agradezco haber nacido en Cuba, de lo contrario no hubiera salvado mi vida</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169969.jpg" class="img-responsive" alt="La recuperación de Yaquelín, una proeza de la ciencia médica cubana" >
    </figure>
    <div class="sumario">
        <p>La enfermera internacionalista que padeció en formas muy graves la COVID-19 y mantuvo en vilo al país, ya está de vuelta a la vida y a su hogar</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-22/agradezco-haber-nacido-en-cuba-de-lo-contrario-no-hubiera-salvado-mi-vida-22-05-2020-22-05-09#comentarios">13 COMENTARIOS</a></p>
			</article>                    <article id="story_4526677" data-id="4526677" data-image="169985">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-23/barcos-iranies-asisten-al-velorio-de-james-monroe-23-05-2020-00-05-45">Barcos iraníes asisten al velorio de James Monroe</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169985.jpg" class="img-responsive" alt="El mundo sigue con atención el avance de los barcos de la nación persa hacia Venezuela. Foto: AFP
" >
    </figure>
    <div class="sumario">
        <p>La trama del valiente gesto solidario iraní con Venezuela se desenvuelve en un escenario cargado de tensiones y peligros</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-23/barcos-iranies-asisten-al-velorio-de-james-monroe-23-05-2020-00-05-45#comentarios">12 COMENTARIOS</a></p>
			</article>                    <article id="story_4526679" data-id="4526679" data-image="163205">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-23/reino-unido-a-la-ue-recapaciten-o-no-habra-acuerdo-post-brexit-23-05-2020-00-05-57">Reino Unido a la UE: recapaciten o no habrá acuerdo post-Brexit</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/03/thumb/f0163205.jpg" class="img-responsive" alt="Londres amenaza abandonar negociación posbrexit con la UE" >
    </figure>
    <div class="sumario">
        <p>Desde el pasado 31 de enero, ambas partes trabajan para alcanzar un acuerdo de libre comercio; sin embargo, muy poco se ha progresado en las negociaciones</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-23/reino-unido-a-la-ue-recapaciten-o-no-habra-acuerdo-post-brexit-23-05-2020-00-05-57#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526673" data-id="4526673" data-image="169977">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-22/limpieza-y-desinfeccion-itinerario-seguro-22-05-2020-23-05-16">Limpieza y desinfección, itinerario seguro</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169977.jpg" class="img-responsive" alt="El colectivo de trabajadores del hospital Faustino Pérez asumió la mayor parte de la desinfección y 
limpieza en el menor tiempo posible. Foto del autor " >
    </figure>
    <div class="sumario">
        <p>En el Hospital Faustino Pérez, el orden y la limpieza ofrecen el mejor semblante de un centro asistencial de salud</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-22/limpieza-y-desinfeccion-itinerario-seguro-22-05-2020-23-05-16#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526597" data-id="4526597" data-image="169899">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba/2020-05-21/cuba-estrena-dos-nuevos-canales-hd3-y-hd4-21-05-2020-10-05-07">Nuevos transmisores amplían cobertura de la televisión digital en Cuba y se habilitan dos canales en HD</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169899.jpg" class="img-responsive" alt="Los nuevos centros de monitoreo permiten detectar con mayor agilidad cualquier falla de la señal y actuar de forma oportuna." >
    </figure>
    <div class="sumario">
        <p><span class="tojvnm2t a6sixzi8 abs2jz4q a8s20v7p t1p8iaqh k5wvi7nf q3lfd5jv pk4s997a bipmatt0 cebpdrjk qowsmv63 owwhemhu dp1hu0rb dhp61c6y iyyx5f41">El aumento en la cantidad de canales es posible gracias a la instalación de una nueva «cabecera de línea», la cual posee mayor capacidad y permite crear otras facilidades técnicas, detalló Justo Moreno García, Director General de la Empresa de Radiocomunicación y Difusión de Cuba (Radiocuba)<br></span></p>    </div>
			<p class="g-story-comments"><a href="cuba/2020-05-21/cuba-estrena-dos-nuevos-canales-hd3-y-hd4-21-05-2020-10-05-07#comentarios">93 COMENTARIOS</a></p>
			</article>                    <article id="story_4526589" data-id="4526589" data-image="169847">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba/2020-05-21/veinte-dias-de-silencio-complice-21-05-2020-01-05-55">Veinte días de silencio cómplice</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169847.jpg" class="img-responsive" alt="Reunión de corresponsales del periódico Granma en la provincia de Cienfuegos.  
Visita al municipio Cumanayagua. Montaña del Escambray." >
    </figure>
    <div class="sumario">
        <p>Al cabo de 20 días de la acción terrorista contra la Embajada cubana en Washington, ni el hecho en sí, ni las reiteradas declaraciones del Gobierno de la Isla en demanda de explicaciones oficiales por parte del país donde ocurrió el peligroso suceso, constituyen la noticia más relevante</p>    </div>
			<p class="g-story-comments"><a href="cuba/2020-05-21/veinte-dias-de-silencio-complice-21-05-2020-01-05-55#comentarios">6 COMENTARIOS</a></p>
			</article>                    <article id="story_4526593" data-id="4526593" data-image="126269">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba/2020-05-21/que-nos-recuerda-un-20-de-mayo-21-05-2020-01-05-25">¿Qué nos recuerda un 20 de mayo?</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2018/12/thumb/f0126269.jpg" class="img-responsive" alt="Acto de izar la Bandera en el Palacio de los Generales el 20 de Mayo 1902
Publicada: 21/05/2001

Hist0068" >
    </figure>
    <div class="sumario">
        <p>Un 20 de mayo, pero de 1902, quedó constituida la República. Se proclamaron derechos formales, pero, en efecto, ¿qué significó la República para muchos cubanos?</p>    </div>
			<p class="g-story-comments"><a href="cuba/2020-05-21/que-nos-recuerda-un-20-de-mayo-21-05-2020-01-05-25#comentarios">9 COMENTARIOS</a></p>
			</article>                    <article id="story_4526655" data-id="4526655" data-image="169941">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba/2020-05-22/bandec-dismunuye-tasas-de-interes-a-unidades-productoras-de-alimentos-y-a-agricultores">Bandec disminuye tasas de interés a unidades productoras de alimentos y a agricultores</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169941.jpg" class="img-responsive" alt="Bandec" >
    </figure>
    <div class="sumario">
        <p>Arroz, frijoles, maíz, vegetales y hortalizas, ganado menor y producción porcina, constituyen líneas de marcado interés nacional para cuyo desarrollo Bandec prioriza financiamientos</p>    </div>
			<p class="g-story-comments"><a href="cuba/2020-05-22/bandec-dismunuye-tasas-de-interes-a-unidades-productoras-de-alimentos-y-a-agricultores#comentarios">3 COMENTARIOS</a></p>
			</article>                    <article id="story_4526647" data-id="4526647" data-image="169907">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-22/condenan-inclusion-de-cuba-en-lista-terrorista-de-estados-unidos-22-05-2020-01-05-55">Condenan inclusión de Cuba en lista terrorista de Estados Unidos</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169907.jpg" class="img-responsive" alt="Caricatura Cuba no está sola. bola del mundo" >
    </figure>
    <div class="sumario">
        <p>El Comando Central del Ejército de Liberación Nacional (ELN) reafirmó que es «de máxima perfidia calificar a Cuba como violadora del Derecho Internacional y hacerlo como pago por sus servicios a la paz, cuando los violadores son Trump y Duque»</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-22/condenan-inclusion-de-cuba-en-lista-terrorista-de-estados-unidos-22-05-2020-01-05-55#comentarios">2 COMENTARIOS</a></p>
			</article>                    <article id="story_4526605" data-id="4526605" data-image="169887">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba/2020-05-21/etecsa-responde-a-inquietudes-de-sus-usuarios-relacionados-con-el-consumo-de-sus-servicios-21-05-2020-11-05-17">Etecsa responde a inquietudes de sus usuarios relacionados con el consumo de sus servicios</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169887.jpg" class="img-responsive" alt="Tania Velázquez Rodríguez, su vicepresidenta primera" >
    </figure>
    <div class="sumario">
        <p>La Empresa de Telecomunicaciones de Cuba, Etecsa, aclaró algunas de las inquietudes de la población generadas en las últimas semanas y referidas a irregularidades en el consumo acelerado de servicios populares como Internet en el móvil y Nauta Hogar</p>    </div>
			<p class="g-story-comments"><a href="cuba/2020-05-21/etecsa-responde-a-inquietudes-de-sus-usuarios-relacionados-con-el-consumo-de-sus-servicios-21-05-2020-11-05-17#comentarios">11 COMENTARIOS</a></p>
			</article>        
        </div>

        <div id="column_1" class="col-md-6 col-sm-6 g-regular-story container-right">

                    <article id="story_4526705" data-id="4526705" data-image="165157">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/informacion-minsap/2020-05-23/ministerio-de-salud-publica-cuba-reporta-15-nuevos-casos-de-covid-19-para-un-acumulado-de-1931-confirmados">Ministerio de Salud Pública: Cuba reporta 15 nuevos casos de COVID-19 para un acumulado de 1931 confirmados</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/03/thumb/f0165157.jpg" class="img-responsive" alt="Información del Minsap" >
    </figure>
    <div class="sumario">
        <p>Con la presencia del doctor Francisco Durán García, director de Epidemiología del Minsap, inició la conferencia de prensa sobre la actualización de la situación de la COVID-19 en Cuba</p>    </div>
			<p class="g-story-comments"><a href="informacion-minsap/2020-05-23/ministerio-de-salud-publica-cuba-reporta-15-nuevos-casos-de-covid-19-para-un-acumulado-de-1931-confirmados#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526707" data-id="4526707" data-image="170011">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cultura/2020-05-23/muere-la-destacada-narradora-oral-cubana-haydee-arteaga-23-05-2020-09-05-30">Muere la destacada narradora oral cubana Haydee Arteaga</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0170011.jpg" class="img-responsive" alt="cultura" >
    </figure>
    <div class="sumario">
        <p>La narradora, quien además había estudiado Solfeo y Teoría en el Conservatorio de La Habana, manifestó siempre su inclinación por la narración oral, gracias a las enseñanzas de su abuela, es por ello que desde sus inicios, su trabajo más fuerte y dedicado fue con los más pequeños</p>    </div>
			<p class="g-story-comments"><a href="cultura/2020-05-23/muere-la-destacada-narradora-oral-cubana-haydee-arteaga-23-05-2020-09-05-30#comentarios">2 COMENTARIOS</a></p>
			</article>                    <article id="story_4526709" data-id="4526709" data-image="125843">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-23/garantizo-cuba-llegada-al-pais-de-80-toneladas-de-material-medico-de-donacion-contra-la-covid-19-23-05-2020-10-05-19">Garantizó Cuba llegada al país de 80 toneladas de material médico de donación contra la COVID-19</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2018/12/thumb/f0125843.jpg" class="img-responsive" alt="bandera cuba china" >
    </figure>
    <div class="sumario">
        <p>A través de su cuenta oficial en la red social Twitter, el titular de transporte, Eduardo Rodríguez Dávila, informó que la llegada al país de dichos insumos se realizó bajo las más estrictas medidas sanitarias</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-23/garantizo-cuba-llegada-al-pais-de-80-toneladas-de-material-medico-de-donacion-contra-la-covid-19-23-05-2020-10-05-19#comentarios">1 COMENTARIO</a></p>
			</article>                    <article id="story_4526713" data-id="4526713" data-image="170029">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-23/juez-estadounidense-autoriza-liquidacion-de-refinerias-venezolanas-confiscadas-ilegalmente-por-trump-23-05-2020-11-05-28">Juez estadounidense autoriza liquidación de refinerías venezolanas confiscadas ilegalmente por Trump</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0170029.jpg" class="img-responsive" alt="citgo" >
    </figure>
    <div class="sumario">
        <p>El analista Gustavo Borges afirmó que «la operación Guaidó se diseñó exactamente para esto: saquear y piratear los preciados activos venezolanos en beneficio de los grandes capitales»</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-23/juez-estadounidense-autoriza-liquidacion-de-refinerias-venezolanas-confiscadas-ilegalmente-por-trump-23-05-2020-11-05-28#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526697" data-id="4526697" data-image="170003">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/deportes/2020-05-23/el-movimiento-deportivo-cubano-ha-confirmado-su-esencia-autenticamente-humana-23-05-2020-00-05-26">El movimiento deportivo cubano ha confirmado su esencia auténticamente humana</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0170003.jpg" class="img-responsive" alt="Entre las glorias deportivas cubanas que se unieron desde el principio a la batalla contra la covid-19, Ana Fidelia Quirot tuvo la iniciativa de hacer nasobucos para la población. Foto: Prensa Latina" >
    </figure>
    <div class="sumario">
        <p>Intervino Osvaldo Vento Montiller, presidente del Inder, en evento Iberoamericano del Deporte</p>    </div>
			<p class="g-story-comments"><a href="deportes/2020-05-23/el-movimiento-deportivo-cubano-ha-confirmado-su-esencia-autenticamente-humana-23-05-2020-00-05-26#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526695" data-id="4526695" data-image="167165">
    <h3 class="g-story-epi">historia de los coronavirus</h3>
    <h2><a href="/consejos-covid/2020-05-23/los-primeros-con-corona-23-05-2020-00-05-24">Los primeros con corona</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/04/thumb/f0167165.jpg" class="img-responsive" alt="coronavirus" >
    </figure>
    <div class="sumario">
        <p>El término «coronavirus» fue acuñado en 1968, basado en cómo, bajo un microscopio electrónico, su superficie se parecía a la capa externa del Sol, llamada corona</p>    </div>
			<p class="g-story-comments"><a href="consejos-covid/2020-05-23/los-primeros-con-corona-23-05-2020-00-05-24#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526699" data-id="4526699" data-image="170001">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/conexion-toxio-2020/2020-05-23/japon-centrado-en-los-preparativos-de-tokio-23-05-2020-01-05-44">Japón centrado en los preparativos de Tokio</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0170001.jpg" class="img-responsive" alt="Tokio confía en poder dejar atrás la pandemia para inaugurar sus Juegos Olímpicos el 23 de julio de 2021. Foto: getty images" >
    </figure>
    <div class="sumario">
        <p>Aprobada en marzo la posposición de los Juegos de Tokio para el verano de 2021, los comités olímpicos nacionales expresaron su beneplácito con la medida, que situó en el centro de atención el cuidado de la salud de los deportistas, aficionados y de todas las personas involucradas en la preparación del evento</p>    </div>
			<p class="g-story-comments"><a href="conexion-toxio-2020/2020-05-23/japon-centrado-en-los-preparativos-de-tokio-23-05-2020-01-05-44#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526701" data-id="4526701" data-image="0">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/deportes/2020-05-23/forcejeo-por-rebaja-salarial-en-futbol-de-la-ciudad-luz-23-05-2020-01-05-33">Forcejeo por rebaja salarial en fútbol de la Ciudad Luz</a></h2>
    <div class="sumario">
        <p>Según publicó este viernes el diario L'Equipe, el presidente del club, Nasser Al-Khelaifi, ha elevado el tono después de que muchos de sus jugadores ni siquiera le escuchan para negociar esa reducción de sueldo</p>    </div>
			<p class="g-story-comments"><a href="deportes/2020-05-23/forcejeo-por-rebaja-salarial-en-futbol-de-la-ciudad-luz-23-05-2020-01-05-33#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526671" data-id="4526671" data-image="159807">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cultura/2020-05-22/virus-aislamientolectura-22-05-2020-22-05-30">Virus, aislamiento…lectura</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/01/thumb/f0159807.jpg" class="img-responsive" alt="Concurso Leer a Martí" >
    </figure>
    <div class="sumario">
        <p>Como parte que soy del grupo de riesgo por la edad he encontrado muy beneficioso el consejo de los psicólogos de ponernos metas cada día, pero se va haciendo difícil: horarios para arreglar en la casa lo que ni siquiera hay que ordenar, escuchar música, atender las informaciones, ver la televisión, las series, las películas y aquellas que tenemos en los equipos, les llamo mi particular historia del cine, y naturalmente leer, leer</p>    </div>
			<p class="g-story-comments"><a href="cultura/2020-05-22/virus-aislamientolectura-22-05-2020-22-05-30#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526669" data-id="4526669" data-image="169973">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-22/activa-presencia-juvenil-en-programas-prioritarios-22-05-2020-22-05-58">Activa presencia juvenil en programas prioritarios</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169973.jpg" class="img-responsive" alt=" 	
El contingente juvenil Ernesto Che Guevara cumplirá misiones de carácter social en tierra agramontina." >
    </figure>
    <div class="sumario">
        <p>Contingente estudiantil camagüeyano respalda acciones en el combate contra la COVID-19</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-22/activa-presencia-juvenil-en-programas-prioritarios-22-05-2020-22-05-58#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526683" data-id="4526683" data-image="169983">
    <h3 class="g-story-epi">Hilo Directo</h3>
    <h2><a href="/hilo-directo/2020-05-23/hilo-directo-23-05-2020-00-05-51">Proponen proyecto de ley en EE.UU.; Destacan eficacia del sistema sanitario de Rusia; Alcalde de Manaos acusa al presidente de Brasil; Un avión de Pakistán se precipitó a tierra</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169983.jpg" class="img-responsive" alt="avion de pakistan" >
    </figure>
    <div class="sumario">
        <p>Conozca los hechos que son noticia hoy en nuestra sección</p>    </div>
			<p class="g-story-comments"><a href="hilo-directo/2020-05-23/hilo-directo-23-05-2020-00-05-51#comentarios">0 COMENTARIOS</a></p>
			</article>                    <article id="story_4526623" data-id="4526623" data-image="169911">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-22/celulas-madre-contra-secuelas-de-la-covid-19-en-cuba-22-05-2020-00-05-49">¿Células madre contra secuelas de la COVID-19 en Cuba?</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169911.jpg" class="img-responsive" alt="El ensayo clínico sobre el empleo de células madre adultas en pacientes recuperados que presentan secuelas de la COVID-19, comenzó el 5 de mayo en el Instituto de Hematología e Inmunología. " >
    </figure>
    <div class="sumario">
        <p>El tratamiento en investigación tiene el propósito de eliminar o disminuir las lesiones inflamatorias intersticiales o fibróticas pulmonares posteriores a la infección y prevenir el avance&nbsp; de la enfermedad, mejorando la calidad de vida</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-22/celulas-madre-contra-secuelas-de-la-covid-19-en-cuba-22-05-2020-00-05-49#comentarios">2 COMENTARIOS</a></p>
			</article>                    <article id="story_4526577" data-id="4526577" data-image="169839">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-21/fidel-contra-la-covid-19-y-mas-alla-21-05-2020-00-05-18">Fidel contra la COVID-19 y más allá</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169839.jpg" class="img-responsive" alt="Fidel Castro en el Centro Nacional de Genética Médica, junto al Dr. Juan C. Dupuy Núñez, coordinador fundador del Contingente Médico Internacional para el enfrentamiento a desastres y graves epidemias Henry Reeve. foto: archivo de granma" >
    </figure>
    <div class="sumario">
        <p>Fue a mediados de los años 80 del siglo pasado, cuando Fidel comenzó a hablar, mientras en Cuba se multiplicaban las facultades de Medicina y crecía el número de estudiantes de especialidades relacionadas con la Salud, entre las dudas de no pocos escépticos y las burlas de sus enemigos, de que el país sería una potencia médica mundial</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-21/fidel-contra-la-covid-19-y-mas-alla-21-05-2020-00-05-18#comentarios">8 COMENTARIOS</a></p>
			</article>                    <article id="story_4526663" data-id="4526663" data-image="169949">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/cuba-covid-19/2020-05-22/plataforma-interreligiosa-de-cuba-repudia-hostilidad-de-estados-unidos-contra-la-isla-22-05-2020-12-05-55">Plataforma Interreligiosa de Cuba expresa su indignación por la hostilidad de EE. UU. hacia la Isla</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169949.jpg" class="img-responsive" alt="Plataforma Interreligiosa de Cuba repudia hostilidad de Estados Unidos contra la Isla" >
    </figure>
    <div class="sumario">
        <p>El movimiento, que integra a las siete religiones establecidas en el territorio nacional, expresó su indignación con la «injusta» inclusión de Cuba en la lista de países que «no cooperan con los esfuerzos antiterroristas»</p>    </div>
			<p class="g-story-comments"><a href="cuba-covid-19/2020-05-22/plataforma-interreligiosa-de-cuba-repudia-hostilidad-de-estados-unidos-contra-la-isla-22-05-2020-12-05-55#comentarios">1 COMENTARIO</a></p>
			</article>                    <article id="story_4526661" data-id="4526661" data-image="169945">
    <h3 class="g-story-epi"></h3>
    <h2><a href="/mundo/2020-05-22/se-estrella-avion-en-pakistan-con-99-personas-reportan-un-sobreviviente-videos">Se estrella avión en Pakistán con 107 personas a bordo, reportan un sobreviviente (+Fotos) (+Videos)</a></h2>
    <figure class="col-md-6 g-story-img">
        <img src="/file/img/2020/05/thumb/f0169945.jpg" class="img-responsive" alt="ACCIDENTE AÉREO EN PAKISTÁN" >
    </figure>
    <div class="sumario">
        <p>El primer ministro pakistaní ha anunciado la inmediata apertura de una investigación para aclarar las causas del accidente. Mientras tanto, el portavoz de la PIA, Abdullah H. Khan, en un video comunicado señaló que «lo último que supimos del piloto fue que (el avión)tenía algún problema técnico»</p>    </div>
			<p class="g-story-comments"><a href="mundo/2020-05-22/se-estrella-avion-en-pakistan-con-99-personas-reportan-un-sobreviviente-videos#comentarios">0 COMENTARIOS</a></p>
			</article>        
        </div>
        <div class="clearfix"></div>
        <div>
    <div class="col-md-6 col-sm-6 left-widgets sidebar-widget container-left">
        <a  title="tirandole" href="/tirandole"><img src="/file/banner/tirandole.jpg" class="img-responsive" alt="tirandole"></a>    </div>
    <div class="col-md-6 col-sm-6 left-widgets sidebar-widget container-right">
        <a target="_blank" title="ecured" href="http://www.ecured.cu"><img src="/file/banner/ecured.jpg" class="img-responsive" alt="ecured"></a>    </div>
    <div class="col-md-6 col-sm-6 left-widgets sidebar-widget container-left">
        <div class="category-widget">
    <h3 class="category-widget-header">Mundo</h3>
    <figure>
        <a href="mundo/2020-05-16/insiste-trump-para-que-obama-testifique-ante-el-senado-sobre-el-mayor-crimen-politico-en-eeuu-16-05-2020-10-05-45"><img src="/file/img/2020/05/thumb/f0169469.jpg" class="img-responsive" alt="eeuu" ></a>
    </figure>
    <article class="category-widget-texts">
        <h2><a href="mundo/2020-05-16/insiste-trump-para-que-obama-testifique-ante-el-senado-sobre-el-mayor-crimen-politico-en-eeuu-16-05-2020-10-05-45">Insiste Trump para que Obama testifique ante el Senado «sobre el mayor crimen político» en EE.UU.</a></h2>
        <p>El presidente de la comisión, Lindsey Graham, señaló que tanto Obama como Trump eran bienvenidos a presentarse ante la Comisión del Senado para intercambiar sus preocupaciones, sin embargo, señaló que tenía «grandes dudas sobre si sería prudente para el país»</p>        <p class="g-story-comments"><a href="mundo/2020-05-16/insiste-trump-para-que-obama-testifique-ante-el-senado-sobre-el-mayor-crimen-politico-en-eeuu-16-05-2020-10-05-45#comentarios">5 COMENTARIOS</a></p>
    </article>
</div>    </div>
    <div class="col-md-6 col-sm-6 left-widgets sidebar-widget container-right">
        <div class="category-widget">
    <h3 class="category-widget-header">Cuba frente a la Covid-19</h3>
    <figure>
        <a href="cuba-covid-19/2020-05-16/frente-a-la-covid-19-mas-disciplinados-que-nunca-16-05-2020-00-05-58"><img src="/file/img/2020/05/thumb/f0169459.jpg" class="img-responsive" alt="En la reunión del grupo de trabajo para la prevención y control de la covid-19, el Presidente Díaz-Canel destacó como otro logro de la batalla de Cuba contra la pandemia el hecho de que no hayan colapsado las instituciones médicas. foto:estudios revolución" ></a>
    </figure>
    <article class="category-widget-texts">
        <h2><a href="cuba-covid-19/2020-05-16/frente-a-la-covid-19-mas-disciplinados-que-nunca-16-05-2020-00-05-58">Frente a la COVID-19, más disciplinados que nunca</a></h2>
        <p>En la reunión del grupo de trabajo para la prevención y el control del nuevo coronavirus –dirigida como es habitual por el Presidente de la República, Miguel Díaz-Canel Bermúdez, y el primer ministro, Manuel Marrero Cruz–, el titular de Salud Pública, José Ángel Portal Miranda, detalló la situación en la que se encuentra el país con 1 840 casos confirmados; de ellos, solo 334 se mantienen ingresados</p>        <p class="g-story-comments"><a href="cuba-covid-19/2020-05-16/frente-a-la-covid-19-mas-disciplinados-que-nunca-16-05-2020-00-05-58#comentarios">1 COMENTARIO</a></p>
    </article>
</div>    </div>
</div>  

    </div>

    <div role="navigation" id="sidebar" class="col-md-4 hidden-xs sidebar">
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Cuba frente al Covid-19" href="cuba-covid-19"><img src="/file/banner/covid-19" class="img-responsive" alt="Cuba frente al Covid-19"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Información del Minsap" href="/informacion-minsap"><img src="/file/banner/banner_minsap" class="img-responsive" alt="Información del Minsap"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Consejos y trucos contra la Covid-19" href="/consejos-covid"><img src="/file/banner/banner_trucos_covid19" class="img-responsive" alt="Consejos y trucos contra la Covid-19"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Fake News: La mentira tiene patas cortas" href="fake-news"><img src="/file/banner/banner_fake_news" class="img-responsive" alt="Fake News: La mentira tiene patas cortas"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Conexión Tokio 2020" href="conexion-toxio-2020"><img src="/file/banner/banner_conexion_tokio" class="img-responsive" alt="Conexión Tokio 2020"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <h3 class="sidebar-widget-header">Opiniones</h3>
<ul class="media-list opiniones">
    <li class="media">
            <a class="pull-left" href="/opinion/elson-concepcion-perez">
            <img width="50" height="50" class="media-object" src="/file/avt/hem_918890.jpg" alt="Elson Concepción Pérez">
        </a>
        <div class="media-body">
            <h4 class="media-heading"><a href="opinion/2020-05-23/cronica-para-un-doble-9-23-05-2020-00-05-58">Crónica para un «doble 9»</a></h4>
            <p>Elson Concepción Pérez | 23 de mayo de 2020</p>
             <p class="g-story-comments"><a href="opinion/2020-05-23/cronica-para-un-doble-9-23-05-2020-00-05-58#comentarios">0 COMENTARIOS</a></p>
        </div>
    </li>
    <li class="media">
            <a class="pull-left" href="/opinion/laura-mercedes-giraldez">
            <img width="50" height="50" class="media-object" src="/file/avt/hem_2ba869.jpg" alt="Laura Mercedes Giráldez">
        </a>
        <div class="media-body">
            <h4 class="media-heading"><a href="opinion/2020-05-23/reinventarse-23-05-2020-00-05-57">Reinventarse</a></h4>
            <p>Laura Mercedes Giráldez | 23 de mayo de 2020</p>
             <p class="g-story-comments"><a href="opinion/2020-05-23/reinventarse-23-05-2020-00-05-57#comentarios">1 COMENTARIO</a></p>
        </div>
    </li>
    <li class="media">
            <a class="pull-left" href="/opinion/marta-valdes">
            <img width="50" height="50" class="media-object" src="/file/avt/hem_8abfe8.jpg" alt="Marta Valdés">
        </a>
        <div class="media-body">
            <h4 class="media-heading"><a href="opinion/2020-05-23/retrato-hablado-23-05-2020-00-05-31">Retrato hablado</a></h4>
            <p>Marta Valdés | 23 de mayo de 2020</p>
             <p class="g-story-comments"><a href="opinion/2020-05-23/retrato-hablado-23-05-2020-00-05-31#comentarios">1 COMENTARIO</a></p>
        </div>
    </li>
</ul>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="discursos-de-diaz-canel" href="/discursos-de-diaz-canel"><img src="/file/banner/discursos-diaz-canel.jpg" class="img-responsive" alt="Banner"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <div class="sidebar-widget discursos-raul">
    <a href="/discursos-raul" title="Ver todos los discursos">
        <img class="img-responsive" src="/file/banner/discursos-raul.jpg" alt="Discursos de Raúl">
    </a>
    <h4><a href="/discursos-raul/2019-08-06/estados-unidos-plantea-la-amenaza-mas-perentoria-de-las-ultimas-cinco-decadas-a-la-paz-en-la-region-06-08-2019-22-08-19">Estados Unidos plantea la amenaza más perentoria de las últimas cinco décadas a la paz en la región</a></h4>
</div>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Fidel" href="http://www.granma.cu/fidel"><img src="/file/banner/banner-fidel-1.jpg" class="img-responsive" alt="Banner"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <h3 class="sidebar-widget-header">Grupos de WhatsApp y Telegram del Periódico Granma</h3>

<!--para Youtube con los relacionados quitadosss-->
<iframe src="https://www.youtube.com/embed/TsG61Gwddc4?rel=0" frameborder="0" allowfullscreen="allowfullscreen"></iframe>

        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Entre líneas" href="entre-lineas"><img src="/file/banner/banner_entre_lineas" class="img-responsive" alt="Entre líneas"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Fotorreportajes" href="/fotorreportajes"><img src="/file/banner/banner_fotorreportajes.jpg" class="img-responsive" alt="Fotorreportajes"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Videos de Granma" href="videos-de-granma"><img src="/file/banner/banner_videos" class="img-responsive" alt="Videos de Granma"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <div class="sidebar-widget cartas-redaccion">
    <a href="/cartas"><img class="img-responsive" src="/file/banner/banner-cartas.jpg" alt="Cartas a la dirección"></a>
    <h4><a href="/cartas/2020-05-22/pide-orientacion-para-el-habitable">Pide orientación para el habitable</a></h4>
</div>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Titulares de Granma en tu móvil" href="/cuba/2014-12-10/titulares-de-granma-en-tu-movil"><img src="/file/banner/Titular en tu movil.jpg" class="img-responsive" alt="Titulares de Granma en tu móvil"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a target="_blank" title="Helms-Burton" href="/helms-burton"><img src="/file/banner/hELMS-b.jpg" class="img-responsive" alt="Helms-Burton"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="hilo directo" href="/hilo-directo"><img src="/file/banner/Hilo 2.jpg" class="img-responsive" alt="Banner"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <div class="category-widget">
    <h3 class="category-widget-header">Reforma Constitucional en Cuba</h3>
    <figure>
        <a href="reforma-constitucional/2019-01-05/en-pdf-nueva-constitucion-de-la-republica-de-cuba"><img src="/file/img/2019/01/thumb/f0127645.jpg" class="img-responsive" alt="constitución" ></a>
    </figure>
    <article class="category-widget-texts">
        <h2><a href="reforma-constitucional/2019-01-05/en-pdf-nueva-constitucion-de-la-republica-de-cuba">En PDF, nueva Constitución de la República de Cuba</a></h2>
        <p>En el transcurso de la próxima semana, Correos de Cuba pondrá a la venta en todas sus unidades y estanquillos de prensa, la Constitución de la República de Cuba que fuera aprobada en el Segundo Período Ordinario de Sesiones de la IX Legislatura de la Asamblea Nacional del Poder Popular, al precio de un peso en moneda nacional</p>        <p class="g-story-comments"><a href="reforma-constitucional/2019-01-05/en-pdf-nueva-constitucion-de-la-republica-de-cuba#comentarios">44 COMENTARIOS</a></p>
    </article>
</div>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="" href="/hoy-en-la-historia"><img src="/file/banner/hoy-en-la-historia.jpg" class="img-responsive" alt="Banner"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Pensamiento" href="/pensamiento"><img src="/file/banner/Pensamiento.jpg" class="img-responsive" alt="Pensamiento"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Entérese" href="/enterese"><img src="/file/banner/enterate.jpg" class="img-responsive" alt="Entérese"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="banner_programacion-cine" href="/programacion-cinematografica"><img src="/file/banner/banner_programacion-cine.jpg" class="img-responsive" alt="banner_programacion-cine"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a target="_blank" title="Ctrl + F" href="/ctrl-f"><img src="/file/banner/banner ciencias-01.jpg" class="img-responsive" alt="Banner"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <h3 class="sidebar-widget-header">BLOGS DE LOS PERIODISTAS</h3>
<ul class="media-list blogs-periodistas">
    <li class="media">
        <a class="pull-left" target="_blank" href="http://lavinadeloslumiere.blogspot.com/2020/03/un-mundo-sin-pantallas_16.html/03/un-mundo-sin-pantallas_16.html">
            <img width="50" height="50" class="media-object" src="/file/avt/hem_136f95.jpg" alt="Julio Martínez Molina">
        </a>
        <div class="media-body">
            <p>Julio Martínez Molina</p>
            <h4 class="media-heading"><a target="_blank" href="http://lavinadeloslumiere.blogspot.com/2020/05/los-corona-malthusianistas.html">Los corona-malthusianistas</a></h4>
        </div>
    </li>
    <li class="media">
        <a class="pull-left" target="_blank" href="http://lavinadeloslumiere.blogspot.com/2020/03/un-mundo-sin-pantallas_16.html/03/un-mundo-sin-pantallas_16.html">
            <img width="50" height="50" class="media-object" src="/file/avt/hem_136f95.jpg" alt="Julio Martínez Molina">
        </a>
        <div class="media-body">
            <p>Julio Martínez Molina</p>
            <h4 class="media-heading"><a target="_blank" href="http://lavinadeloslumiere.blogspot.com/2020/04/la-llama-encendida-por-sanders.html">La llama encendida por Sanders</a></h4>
        </div>
    </li>
    <li class="media">
        <a class="pull-left" target="_blank" href="http://lavinadeloslumiere.blogspot.com/?m=1">
            <img width="50" height="50" class="media-object" src="/file/avt/hem_013d40.jpg" alt="Julio Martínez Molina">
        </a>
        <div class="media-body">
            <p>Julio Martínez Molina</p>
            <h4 class="media-heading"><a target="_blank" href="http://lavinadeloslumiere.blogspot.com/2020/02/elisa-y-marcela-marianne-y-heloise-jean.html?m=1">Elisa y Marcela, Marianne y Héloïse, Jean y Lydia</a></h4>
        </div>
    </li>
</ul>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Economía con tinta" href="/economia-con-tinta"><img src="/file/banner/bannereconomíacontinta.jpg" class="img-responsive" alt="Economía con tinta"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a target="_blank" title="documentos-del-7mo-congreso1" href="/file/pdf/gaceta/último PDF 32.pdf"><img src="/file/banner/banner-documentos-del-7mo-congreso1.jpg" class="img-responsive" alt="documentos-del-7mo-congreso1"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a target="_blank" title="" href="/file/pdf/gaceta/tabloide 2 último.pdf"><img src="/file/banner/banner-lineamientos3" class="img-responsive" alt="Banner"></a>        </div>
        </div><!--/sidebar-->

</div>

<div id="s_bottom" class="row especiales-banners">

</div>
<div class="row home-media">

    <div class="col-md-6 container-left">
        <h3 class="especiales-header">La foto del día</h3>
        <a href="/multimedia/imagenes/169585">
            <img src="/file/img/2020/05/thumb/f0169585.jpg" class="img-responsive" alt="Obra de Ángel Luis Velázquez." >
        </a>
        <div class="home-media-caption-bg"></div>
        <div class="home-media-caption">
                    <p>Obra de Ángel Luis Velázquez. <strong>Foto: </strong>Desconocido </p>
                </div>
    </div>

    <div class="col-md-6 container-right">
        <h3 class="especiales-header">Opinión gráfica</h3>
        <a href="/multimedia/imagenes/167989">
            <img src="/file/img/2020/04/thumb/f0167989.jpg" class="img-responsive" alt="Nasobuco" >
        </a>
        <div></div> <!-- class="home-media-caption-bg" class="home-media-caption" -->
        <div class="home-media-caption">
       <!--              <p> <strong>Foto: </strong>Martirena </p>
         -->
        </div>
    </div>

</div>
<div class="row servicios-home">

    <div class="col-md-4 container-left">
        <a href="/cartelera"><img src="/static/img/cartelera.jpg" class="img-responsive" alt="Cartelera" ></a>
    </div>

    <div class="col-md-4 container-center">
        <a href="/portadas-especiales"><img src="/static/img/especiales.jpg" class="img-responsive" alt="Especiales" ></a>
    </div>

    <div class="col-md-4 container-right">
        <a target="_blank" href="http://www.insmet.cu/asp/genesis.asp?TB0=PLANTILLAS&TB1=INICIAL"><img src="/static/img/el-tiempo.jpg" class="img-responsive" alt="El tiempo" ></a>
    </div>

</div>
<div class="row servicios-home">

    <div class="col-md-4 container-left">
        <a href="/cuestion-de-leyes"><img src="/static/img/banner-cuestion-de-leyes.jpg" class="img-responsive" alt="Cuestión de Leyes" ></a>
    </div>

    <div class="col-md-4 container-center">
        <a href="/el-espanol-nuestro"><img src="/static/img/banner-elespanol.png" class="img-responsive" alt="El español" ></a>
    </div>

    <div class="col-md-4 container-right">
        <a href="/todo-salud"><img src="/static/img/todo-salud-banner.jpg" class="img-responsive" alt="Consulta médica" ></a>
    </div>

</div>


        <div id="footer">

            <nav class="navbar navbar-default g-menus g-main-menu g-main-menu-footer hidden-xs" role="navigation">

    <div class="collapse navbar-collapse navbar-main-collapse">

        <ul class="nav navbar-nav">
          <li class="active"><a href="/">Portada</a></li>
                  <li ><a href="/cuba">Cuba</a></li>
                  <li ><a href="/mundo">Mundo</a></li>
                  <li ><a href="/deportes">Deportes</a></li>
                  <li ><a href="/cultura">Cultura</a></li>
                  <li ><a href="/opinion">Opinión</a></li>
                  <li ><a href="/ciencia">Ciencia</a></li>
                  <li ><a href="/salud">Salud</a></li>
                </ul>

        <ul class="nav navbar-nav navbar-right">
          <li><a href="/quienes-somos">¿Quiénes somos?</a></li>
        </ul>

    </div>
</nav>
            <div class="row g-footer-directory">
                <h3>DIRECTORIO DE PRENSA:</h3>
                <div class="col-lg-4 g-footer-directories">
                    <h4>PUBLICACIONES EN INTERNET</h4>
                    <p>
                        <a target="_blank" href=" http://www.juventudrebelde.cu">Juventud Rebelde</a> |
                        <a target="_blank" href="http://www.trabajadores.cu">Trabajadores.</a> |
                        <a target="_blank" href="http://www.cubadebate.cu/">Cubadebate</a> |
                        <a target="_blank" href="http://www.acn.cu">ACN</a> |
                        <a target="_blank" href="http://prensa-latina.cu/">Prensa Latina</a> |
                        <a target="_blank" href="http://www.opciones.cu/">Opciones</a> |
                        <a target="_blank" href="http://cubasi.cu/">CubaSi</a> |
                        <a target="_blank" href="http://www.cubahora.cu/">Cubahora</a> |
                        <a target="_blank" href="http://www.cubaperiodistas.cu/">Cubaperiodistas</a> |
                        <a target="_blank" href="http://www.jit.cu/">JIT</a>
                    </p>
                </div>
                <div class="col-lg-4 g-footer-directories">
                    <h4>REVISTAS</h4>
                    <p>
                        <a target="_blank" href="http://bohemia.cu/">Bohemia</a> |
                        <!--a target="_blank" href="http://www.cubasocialista.cu/">Cuba Socialista</a--> 
                        <a target="_blank" href="http://www.mujeres.co.cu/">Mujeres</a> |
                        <!--a target="_blank" href="http://www.tricontinental.cu/">Tricontinental</a-->
                        <a target="_blank" href="http://www.lajiribilla.cu/">La Jiribilla</a> |
                        <a target="_blank" href="http://www.almamater.cu/">Alma Mater</a> |
                        <a target="_blank" href="http://www.caimanbarbudo.cu/">Caimán Barbudo</a> |
                        <a target="_blank" href="http://www.juventudtecnica.cu/">Juventud Técnica</a> |
                        <a target="_blank" href="http://www.pionero.cu/">Pionero</a>
                    </p>
                </div>
                <div class="col-lg-4 g-footer-directories">
                    <h4>TV Y RADIO</h4>
                    <p>
                        <a target="_blank" href="http://www.tvcubana.icrt.cu/">TV Cubana</a> |
                        <a target="_blank" href="http://www.radiocubana.cu/">Radio Cubana</a> |
                        <a target="_blank" href="http://www.radiohc.cu/">Radio Habana Cuba</a> |
                        <a target="_blank" href="http://www.radioprogreso.cu/">Radio Progreso</a> |
                        <a target="_blank" href="http://www.radiorebelde.cu/">Radio Rebelde</a> |
                        <a target="_blank" href="http://www.radioreloj.cu/ ">Radio Reloj</a> |
                        <a target="_blank" href="http://www.radiotaino.cu/">Radio Taino</a> |
                        <a target="_blank" href="http://www.cmbfradio.cu/">Radio Musical Nacional</a>
                    </p>
                </div>
                 <div class="col-lg-4 g-footer-directories">
                    <h4>ENLACES DE INTER&Eacute;S</h4>
                    <p>
                        <a target="_blank" href="http://www.pcc.cu/">Partido Comunista de Cuba</a> |
                        <a target="_blank" href="http://www.parlamentocubano.cu/">Sitio oficial del Gobierno de la Rep&uacute;blica de Cuba</a> |
                        <a target="_blank" href="http://www.minrex.gob.cu/">Ministerio de Relaciones Exteriores</a> |
                        <a target="_blank" href="http://www.aduana.gob.cu/">Aduana</a>
                    </p>
                </div>
            </div>
            <div class="g-footer-staff">
                <div class="g-footer-logo"><img src="/static/img/granma-logo.png" class="img-responsive" alt="Responsive image" width="100"></div>
                <h5 class="g-footer-subtitle">&Oacute;RGANO OFICIAL DEL COMIT&Eacute; CENTRAL DEL PARTIDO COMUNISTA DE CUBA</h5>
                <h3>GRANMA © 2018 | Fundado el 3 de octubre de 1965 | ISNN 0864-0424 | General Su&aacute;rez y Territorial, Plaza de la Revoluci&oacute;n, La Habana. Cuba | Tel&eacute;fono: 881-3333</h3>
            </div>

        </div>

    </div> <!-- /container -->
	<div class="back-top btn btn-danger">
		<span class="glyphicon glyphicon-chevron-up"></span>
	</div>
    <!--script src="/static/js/jquery.js"></script-->
	<script src="/static/js/linea/jquery2.min.js"></script>

    <script src="/static/js/bootstrap.min.js"></script>
    <script src="/static/js/encuestas.votar.js"></script>
	<script src="/static/js/infografia.js"></script>
	<script async src="https://platform.twitter.com/widgets.js" charset="utf-8"></script>
	<script async src="//www.instagram.com/embed.js"></script>
	
    <script  src="/static/js/linea/index.js"></script>
	<script  src="/static/js/underscore-1.8.3.min.js"></script>
    <script  src="/static/js/quizzer.js"></script>
    

<!-- Google Analytic -->
<script type="text/javascript">

  var _gaq = _gaq || [];
  _gaq.push(['_setAccount', 'UA-112047104-1']);
  _gaq.push(['_trackPageview']);

  (function() {
    var ga = document.createElement('script'); 
	ga.type = 'text/javascript'; 
	ga.async = true;
    ga.src = ('https:' == document.location.protocol ? 'https://ssl' : 'http://www') + '.google-analytics.com/ga.js';
    var s = document.getElementsByTagName('script')[0]; 
	s.parentNode.insertBefore(ga, s);
  })();

</script>
<!-- End Google Analytic -->

<!-- Piwik -->
<script type="text/javascript">
   var _paq = _paq || [];
   _paq.push(["setDocumentTitle", document.domain + "/" + document.title]);
   _paq.push(["setCookieDomain", "*.www.granma.cu"]);
   _paq.push(["setDomains", ["*.www.granma.cu"]]);
   _paq.push(["trackPageView"]);
   _paq.push(["enableLinkTracking"]);

   (function() {
     var u=(("https:" == document.location.protocol) ? "https" : "http") 
+ "://piwikstats.cip.cu/";
     _paq.push(["setTrackerUrl", u+"piwik.php"]);
     _paq.push(["setSiteId", "101"]);
     var d=document, g=d.createElement("script"),
s=d.getElementsByTagName("script")[0]; g.type="text/javascript";
     g.defer=true; g.async=true; g.src=u+"piwik.js";
s.parentNode.insertBefore(g,s);
   })();
</script>
<!-- End Piwik Code -->




</body>
</html>